// Author: Elin.Liu
// Date: 2022-11-25 21:09:22
// Last Modified by:   Elin.Liu
// Last Modified time: 2022-11-25 21:09:22

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>


/*cpu 中的矩阵乘法*/
__host__ void some_func(int *a, int *b, int *c)
{
    int i;

    for (i = 0; i < 128; i++)
    {
        a[i] = b[i] * c[i];
    }
}

/*gpu 中的矩阵乘法*/
__global__ void some_kernel_func(int *a, int *b, int *c)
{
    // 初始化线程ID
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    // 对数组元素进行乘法运算
    a[i] = b[i] * c[i];
    // 打印打前处理的进程ID
    // 可以看到blockIdx并非是按照顺序启动的，这也说明线程块启动的随机性
    printf("blockIdx.x = %d,blockDimx.x = %d,threadIdx.x = %d\n", blockIdx.x, blockDim.x, threadIdx.x);
}

int main(void)
{
    // 初始化指针元素
    int *a, *b, *c;
    // 初始化GPU指针元素
    int *gpu_a, *gpu_b, *gpu_c;
    // 初始化数组大小
    int size = 128 * sizeof(int);
    // 为CPU指针元素分配内存
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);
    // 为GPU指针元素分配内存
    hipMalloc((void **)&gpu_a, size);
    hipMalloc((void **)&gpu_b, size);
    hipMalloc((void **)&gpu_c, size);
    // 初始化数组元素
    for (int i = 0; i < 128; i++)
    {
        b[i] = i;
        c[i] = i;
    }
    // 将数组元素复制到GPU中
    hipMemcpy(gpu_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c, c, size, hipMemcpyHostToDevice);
    // 执行GPU核函数
    some_kernel_func<<<4, 32>>>(gpu_a, gpu_b, gpu_c);
    // 将GPU中的结果复制到CPU中
    hipMemcpy(a, gpu_a, size, hipMemcpyDeviceToHost);
    // 释放GPU和CPU中的内存
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);
    free(a);
    free(b);
    free(c);
    return 0;
}